#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define GRID_SIZE 100
#define BLOCK_SIZE 100

#define MAX_INPUT_COUNT 10
#define MAX_OUTPUT_COUNT 10
#define MAX_WORD_COUNT 20
#define NUM_KEYS 100 
typedef struct tagWord {
	char szWord[30];
}Word;
typedef struct tagKeyPair {
	char szWord[30];
	int nCount;
}KeyPair;

typedef struct tagInputData {
	Word pWord[20];
	int nWordCount;
}InputData;
typedef KeyPair OutputData;

typedef struct tagKeyValueData {
	KeyPair pKeyPair[NUM_KEYS];	
	int nCount;
}KeyValueData;

int g_nKeyData = 0;

__device__ void mapper(InputData *input, KeyValueData *keyData)
{
	int nWordCount = input->nWordCount;
	keyData->nCount = nWordCount;
	for (int i = 0; i<nWordCount; i++) {
		KeyPair* keyPair = &keyData->pKeyPair[i];
		int j = 0;		

		char* p = input->pWord[i].szWord;
		while(*p != 0)
		{ 
			keyPair->szWord[j] = *p;
			p++;
			j++;
		}		
		keyPair->nCount = 1;
	}
}


__device__ bool compare(char* sz1, char* sz2) {
	char* p2 = sz2;
	int n = 0;
	while (*p2 != 0) {
		if (sz1[n] != *p2)
			return false;
		p2++;
		n++;
	}
	return true;
}

__device__ void reducer(KeyValueData *keyData, OutputData *output, int nInputCount, int* nOutputCount)
{
	int nIndex = 0;
	for (int index = 0; index < nInputCount; index++) {
		KeyValueData* pKeyData = &keyData[index];
		int nCount = pKeyData->nCount;
		for (int i = 0; i<nCount; i++) {

			bool bExist = false;
			int j = 0;

			KeyPair keyPair = pKeyData->pKeyPair[i];
			for (int j = 0; j <= nIndex; j++)
			{
				if (compare(output[j].szWord, keyPair.szWord))
				{
					output[j].nCount++;
					bExist = true;
					break;
				}
			}
			if (bExist)
				continue;

			j = 0;

			char* p = keyPair.szWord;
			//printf("Word2:%s\n", keyPair.szWord);
			while (*p != 0)
			{
				output[nIndex].szWord[j] = *p;
				p++;
				j++;
			}
			output[nIndex].nCount = 1;
			nIndex++;
		}
	}
	*nOutputCount = nIndex;
}
__global__ void mapKernel(InputData *input, int nInputCount, KeyValueData *pairs)
{
	int indexWithinTheGrid = threadIdx.x + blockIdx.x * blockDim.x;
	int gridStride = gridDim.x * blockDim.x;
	for (int i = indexWithinTheGrid; i < nInputCount; i += gridStride)
	{
		mapper(&input[i], &pairs[i]);
	}
}

__global__ void reduceKernel(KeyValueData *pairs, int nInputCount, OutputData *output, int* nOutputCount) {
	for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < 1; i += blockDim.x * gridDim.x) {
		reducer(pairs, output, nInputCount, nOutputCount);
	}
}
void cudaMap(InputData *input, int nInputCount, KeyValueData *pairs) {
	mapKernel << <GRID_SIZE, BLOCK_SIZE >> >(input, nInputCount, pairs);

}

void cudaReduce(KeyValueData *pairs, int nInputCount, OutputData *output, int* nOutputCount) {
	reduceKernel << <GRID_SIZE, BLOCK_SIZE >> >(pairs, nInputCount, output, nOutputCount);

}
void runMapReduce(InputData *input, int nInputCount, OutputData *output, int* nOutputCount) {
	InputData   *dev_input;
	OutputData  *dev_output;
	KeyValueData *dev_pairs;
	int			*dev_count;
	size_t input_size = nInputCount * sizeof(InputData);
	size_t output_size = MAX_OUTPUT_COUNT * sizeof(OutputData);
	size_t pairs_size = nInputCount * sizeof(KeyValueData);
	

	hipMalloc(&dev_input, input_size);
	hipMalloc(&dev_pairs, pairs_size);
	hipMalloc(&dev_count, sizeof(int));

	hipMemcpy(dev_input, input, input_size, hipMemcpyHostToDevice);

	cudaMap(dev_input, nInputCount, dev_pairs);

	hipFree(dev_input);

	hipMalloc(&dev_output, output_size);
	hipMemset(dev_output, 0, output_size);
	cudaReduce(dev_pairs, nInputCount, dev_output, dev_count);

	hipMemcpy(output, dev_output, output_size, hipMemcpyDeviceToHost);
	hipMemcpy(nOutputCount, dev_count, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_count);
	hipFree(dev_pairs);
	hipFree(dev_output);
}

int main(int argc, char const *argv[])
{
	printf("Input Data:\n");
	FILE* pFile = fopen("test.txt", "rt");
	char c;
	// Input Text and Splitting
	InputData* pInputData = (InputData*)malloc(MAX_INPUT_COUNT*sizeof(InputData));
	OutputData* pOutputData = (OutputData*)malloc(MAX_OUTPUT_COUNT * sizeof(OutputData));
	int nInputCount = 0;
	int nOutputCount = 0;
	if (pFile) {
		char szLine[100];
		while(!feof(pFile))
		{
			memset(szLine, 0, 100);
			fgets(szLine, 100, pFile);
			printf("%s", szLine);
			pInputData[nInputCount].nWordCount = 0;
			int nWordIndex = 0;
			int nIndex = 0;
			char szWord[20];
			if(strlen(szLine)<2)
				break;
			for (int i = 0; i < strlen(szLine); i++) {				
				if (szLine[i] == ' ' || szLine[i] == 0x0d || szLine[i] == 0x0A)
				{
					szWord[nIndex] = 0;
					nIndex = 0;
					strcpy(pInputData[nInputCount].pWord[nWordIndex].szWord, szWord);
					pInputData[nInputCount].nWordCount++;					
					nWordIndex++;
					if (szLine[i] == 0x0d)
						break;
				}
				else {
					szWord[nIndex] = szLine[i];
					nIndex++;
				}
			}
			nInputCount++;
		}
		fclose(pFile);
	}
	// Splitting End
	int nTotalCount = 0;
	runMapReduce(pInputData, nInputCount, pOutputData, &nOutputCount);
	printf("--------------------------\n");
	
	for (int i = 0; i < nOutputCount; i++) {
		printf("%s\t%d\n", pOutputData[i].szWord, pOutputData[i].nCount);
		nTotalCount += pOutputData[i].nCount;
	}
	// Output WordCount	
	printf("--------------------------\n");
	printf("Total Count:%d\n", nTotalCount);
	free(pOutputData);
	free(pInputData);
	return 0;
}
